#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void compute_accel(vector3* values, vector3** accels, vector3* dVel, vector3* dPos, double* dMass) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= NUMENTITIES || y >= NUMENTITIES) {
		return;
	}
	if (x == 0) {
		accels[x] = &values[x * NUMENTITIES];
	}
	__syncthreads();
	if (x == y) {
		FILL_VECTOR(accels[x][y], 0, 0, 0);
	} else {
		vector3 distance;
		int k;
		for (k = 0; k<3; k++) {
			distance[k] = dPos[x][k] - dPos[y][k];
		};
		double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
		double magnitude = sqrt(magnitude_sq);
		double accelmag = -1 * GRAV_CONSTANT * dMass[y] / magnitude_sq;
		FILL_VECTOR(accels[x][y], accelmag * distance[0] / magnitude, accelmag * distance[1] / magnitude, accelmag * distance[2] / magnitude);
	}
	dPos[x][0] = 1000000000.0;

}

__global__ void add_accel(vector3* values, vector3** accels, vector3* dVel, vector3* dPos, double* dMass) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= NUMENTITIES) {
		return;
	}
	vector3 accel_sum = {0, 0, 0};
	int j, k;
	for (j = 0; j < NUMENTITIES; j++) {
		for (k = 0; k < 3; k++) {
			accel_sum[k] += accels[x][j][k];
		}
	}
	//compute the new velocity based on the acceleration and time interval
	//compute the new position based on the velocity and time interval
	for (k = 0; k < 3; k++) {
		dVel[x][k] += accel_sum[k] * INTERVAL;
		dPos[x][k] = dVel[x][k] * INTERVAL;
	}
	dPos[x][0] = 1000000000.0;
}

void compute() {
	//d_hvel and d_hpos hold the hVel and hPos variables on the GPU
	vector3 *dVel, *dPos;
	double *dMass;
	vector3* dValue;
	vector3** dAccel;

	int e = 0;

	e+=hipMallocManaged(&dVel, (sizeof(vector3) * NUMENTITIES));
	hipMallocManaged(&dPos, (sizeof(vector3) * NUMENTITIES));
	hipMallocManaged(&dMass, (sizeof(double) * NUMENTITIES));

	//Copy memory from the host onto the GPU
	hipMemcpy(dVel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(dPos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(dMass, mass, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);

	//Allocate space on the GPU for these variables
	hipMallocManaged(&dValue, sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	hipMallocManaged(&dAccel, sizeof(vector3*) * NUMENTITIES);

	//Determine number of blocks that we should be running
	dim3 threadsPerBlock(16, 16, 1);
	dim3 numBlocks((NUMENTITIES + 15) / 16, (NUMENTITIES + 15) / 16, 1);

	compute_accel<<<numBlocks, threadsPerBlock>>>(dValue, dAccel, dVel, dPos, dMass);
	hipDeviceSynchronize();

	add_accel<<<(NUMENTITIES + 15) / 16, 16>>>(dValue, dAccel, dVel, dPos, dMass);
	hipDeviceSynchronize();

	//Copy the results back to the device
	hipMemcpy(hVel, dVel, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hPos, dPos, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(mass, dMass, sizeof(double) * NUMENTITIES, hipMemcpyDeviceToHost);

	hipFree(dMass);
	hipFree(dVel);
	hipFree(dPos);
	hipFree(dValue);
	hipFree(dAccel);

	if (e) {
		printf("%d\n", e);
	exit(0);
	}
}
